#include "hip/hip_runtime.h"
﻿//==================================================================================================
// Written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is distributed
// without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication along
// with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
//==================================================================================================
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <stdexcept>


#include <cstdio>
#include <cstdlib>

#include "raytracing.h"

#include "Vec3.h"
#include "Camera.h"
#include "Object.h"
#include "Scene.h"
#include "Sphere.h"
#include "Diffuse.h"
#include "Metallic.h"
#include "Crystalline.h"

#include "random.h"
#include "utils.h"
#include "main.h"

Scene loadObjectsFromFile(const std::string& filename) {
	std::ifstream file(filename);
	std::string line;

	Scene list;

	if (file.is_open()) {
		while (std::getline(file, line)) {
			std::stringstream ss(line);
			std::string token;
			std::vector<std::string> tokens;

			while (ss >> token) {
				tokens.push_back(token);
			}

			if (tokens.empty()) continue;

			// Esperamos al menos la palabra clave "Object"
			if (tokens[0] == "Object" && tokens.size() >= 12) { // M�nimo para Sphere y un material con 1 float
				// Parsear la esfera
				if (tokens[1] == "Sphere" && tokens[2] == "(" && tokens[7] == ")") {
					try {
						float sx = std::stof(tokens[3].substr(tokens[3].find('(') + 1, tokens[3].find(',') - tokens[3].find('(') - 1));
						float sy = std::stof(tokens[4].substr(0, tokens[4].find(',')));
						float sz = std::stof(tokens[5].substr(0, tokens[5].find(',')));
						float sr = std::stof(tokens[6]);

						// Parsear el material del �ltimo objeto creado

						if (tokens[8] == "Crystalline" && tokens[9] == "(" && tokens[11].back() == ')') {
							float ma = std::stof(tokens[10]);
							list.add(new Object(
								new Sphere(Vec3(sx, sy, sz), sr),
								new Crystalline(ma)
							));
							//std::cout << "Crystaline" << sx << " " << sy << " " << sz << " " << sr << " " << ma << "\n";
						}
						else if (tokens[8] == "Metallic" && tokens.size() == 15 && tokens[9] == "(" && tokens[14] == ")") {
							float ma = std::stof(tokens[10].substr(tokens[10].find('(') + 1, tokens[10].find(',') - tokens[10].find('(') - 1));
							float mb = std::stof(tokens[11].substr(0, tokens[11].find(',')));
							float mc = std::stof(tokens[12].substr(0, tokens[12].find(',')));
							float mf = std::stof(tokens[13].substr(0, tokens[13].length() - 1));
							list.add(new Object(
								new Sphere(Vec3(sx, sy, sz), sr),
								new Metallic(Vec3(ma, mb, mc), mf)
							));
							//std::cout << "Metallic" << sx << " " << sy << " " << sz << " " << sr << " " << ma << " " << mb << " " << mc << " " << mf << "\n";
						}
						else if (tokens[8] == "Diffuse" && tokens.size() == 14 && tokens[9] == "(" && tokens[13].back() == ')') {
							float ma = std::stof(tokens[10].substr(tokens[10].find('(') + 1, tokens[10].find(',') - tokens[10].find('(') - 1));
							float mb = std::stof(tokens[11].substr(0, tokens[11].find(',')));
							float mc = std::stof(tokens[12].substr(0, tokens[12].find(',')));
							list.add(new Object(
								new Sphere(Vec3(sx, sy, sz), sr),
								new Diffuse(Vec3(ma, mb, mc))
							));
							//std::cout << "Diffuse" << sx << " " << sy << " " << sz << " " << sr << " " << ma << " " << mb << " " << mc << "\n";
						}
						else {
							std::cerr << "Error: Material desconocido o formato incorrecto en la linea: " << line << std::endl;
						}
					}
					catch (const std::invalid_argument& e) {
						std::cerr << "Error: Conversi�n inv�lida en la linea: " << line << " - " << e.what() << std::endl;
					}
					catch (const std::out_of_range& e) {
						std::cerr << "Error: Valor fuera de rango en la linea: " << line << " - " << e.what() << std::endl;
					}
				}
				else {
					std::cerr << "Error: Formato de esfera incorrecto en la linea: " << line << std::endl;
				}
			}
			else {
				std::cerr << "Error: Formato de objeto incorrecto en la linea: " << line << std::endl;
			}
		}
		file.close();
	}
	else {
		std::cerr << "Error: No se pudo abrir el archivo: " << filename << std::endl;
	}
	return list;
}


Scene randomScene() {
	Scene list;
	list.add(new Object(
		new Sphere(Vec3(0.0f, -1000.0f, 0.0f), 1000.0f),
		new Diffuse(Vec3(0.5f, 0.5f, 0.5f))
	));

	for (int a = -11; a < 11; a++) {
		for (int b = -11; b < 11; b++) {
			float choose_mat = random();
			Vec3 center(a + 0.9f * random(), 0.2f, b + 0.9f * random());
			if ((center - Vec3(4.0f, 0.2f, 0.0f)).length() > 0.9f) {
				if (choose_mat < 0.8f) {  // diffuse
					list.add(new Object(
						new Sphere(center, 0.2f),
						new Diffuse(Vec3(random() * random(),
							random() * random(),
							random() * random()))
					));
				} else if (choose_mat < 0.95f) { // metallic
					list.add(new Object(
						new Sphere(center, 0.2f),
						new Metallic(Vec3(0.5f * (1.0f + random()),
							0.5f * (1.0f + random()),
							0.5f * (1.0f + random())),
							0.5f * random())
					));
				} else {  // crystalline
					list.add(new Object(
						new Sphere(center, 0.2f),
						new Crystalline(1.5f)
					));
				}
			}
		}
	}

	list.add(new Object(
		new Sphere(Vec3(0.0f, 1.0f, 0.0f), 1.0f),
		new Crystalline(1.5f)
	));
	list.add(new Object(
		new Sphere(Vec3(-4.0f, 1.0f, 0.0f), 1.0f),
		new Diffuse(Vec3(0.4f, 0.2f, 0.1f))
	));
	list.add(new Object(
		new Sphere(Vec3(4.0f, 1.0f, 0.0f), 1.0f),
		new Metallic(Vec3(0.7f, 0.6f, 0.5f), 0.0f)
	));

	return list;
}

void rayTracingCPU(Scene world, Vec3* img, int w, int h, int ns = 10) {

	Vec3 lookfrom(13.0f, 2.0f, 3.0f);
	Vec3 lookat(0.0f, 0.0f, 0.0f);
	float dist_to_focus = 10.0f;
	float aperture = 0.1f;

	Camera cam(lookfrom, lookat, Vec3(0.0f, 1.0f, 0.0f), 20.0f, float(w) / float(h), aperture, dist_to_focus);

	for (int j = h - 1; j >= 0; j--) {
		for (int i = 0; i < w; i++) {
			Vec3 col(0.0f, 0.0f, 0.0f);
			for (int s = 0; s < ns; s++) {
				float u = float(i + random()) / float(w);
				float v = float(j + random()) / float(h);
				Ray r = cam.get_ray(u, v);
				col += world.getSceneColor(r);
			}
			col /= float(ns);
			col[0] = sqrt(col[0]);
			col[1] = sqrt(col[1]);
			col[2] = sqrt(col[2]);
			img[j * w + i] = col;
		}
	}
}

int main(int argc, char** argv) {
	////////// width, height, ns, threadsX, threadsY
	int w = std::atoi(argv[1]);
	int h = std::atoi(argv[2]);
	int ns = std::atoi(argv[3]);
	int tx = std::atoi(argv[4]);
	int ty = std::atoi(argv[5]);

	clock_t start, stop;
	double timer_seconds;

	size_t size = sizeof(unsigned char) * w * h * 3;
	unsigned char* data = (unsigned char*)malloc(size);

	Vec3* img;
	size_t isize = w * h * sizeof(Vec3);
	hipMallocManaged((void**)&img, isize);
	/*
	
	Scene world = loadObjectsFromFile("../../../../MPI/Scene1.txt");
	world.setSkyColor(Vec3(0.5f, 0.7f, 1.0f));
	world.setInfColor(Vec3(1.0f, 1.0f, 1.0f));

	std::cerr << "--- CPU ---\n";
	start = clock();
	rayTracingCPU(world, img, w, h, ns);

	for (int i = h - 1; i >= 0; i--) {
		for (int j = 0; j < w; j++) {
			size_t idx = i * w + j;
			data[idx * 3 + 0] = char(255.99 * img[idx].b());
			data[idx * 3 + 1] = char(255.99 * img[idx].g());
			data[idx * 3 + 2] = char(255.99 * img[idx].r());
		}
	}
	stop = clock();
	timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "CPU took " << timer_seconds << " seconds.\n\n";

	writeBMP("imgCPU-prueba.bmp", data, w, h);
	printf("Imagen CPU creada.\n");

	*/
	//std::cerr << "--- GPU ---\n";
	start = clock();
	rayTracingGPU(img, w, h, ns, tx, ty);

	for (int i = h - 1; i >= 0; i--) {
		for (int j = 0; j < w; j++) {
			size_t idx = i * w + j;
			data[idx * 3 + 0] = char(255.99 * img[idx].b());
			data[idx * 3 + 1] = char(255.99 * img[idx].g());
			data[idx * 3 + 2] = char(255.99 * img[idx].r());
		}
	}
	stop = clock();
	timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	//std::cerr << "GPU took " << timer_seconds << " seconds.\n";

	writeBMP("../../../../CUDA/Imagenes/imgGPUImg.bmp", data, w, h);
	//printf("Imagen GPU creada.\n");

	free(data);
	hipDeviceReset();

	// para el CSV
	std::cout << tx << ","
		<< ty << ","
		<< w << ","
		<< h << ","
		<< ns << ","
		<< timer_seconds;
	std::cout << std::endl;

	//getchar();
	return (0);
}