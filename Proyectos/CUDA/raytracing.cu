#include "hip/hip_runtime.h"
﻿//#include <float.h>
#include <cstdio>
//#include <cstdlib>
//#include <limits>

#include <ctime>

#include "Vec3.h"
#include "Ray.h"
#include "Camera.h"
//#include "Object.h"
#include "Scene.h"
#include "Sphere.h"
#include "Diffuse.h"
#include "Metallic.h"
#include "Crystalline.h"

#include "random.h"
//#include "utils.h"


// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int w, int h, hiprandState* rand_state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if ((idx >= w) || (idy >= h)) return;

    int pixel_index = idy * w + idx;
    hiprand_init(42, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(Vec3* fb, int w, int h, int ns, Camera** cam, SceneGPU* world, hiprandState* rand_state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if ((idx >= w) || (idy >= h)) return;

    int pixel_index = idy * w + idx;
    hiprandState local_rand_state = rand_state[pixel_index];
    Vec3 col(0, 0, 0);

    for (int s = 0; s < ns; s++) {
        float u = float(idx + hiprand_uniform(&local_rand_state)) / float(w);
        float v = float(idy + hiprand_uniform(&local_rand_state)) / float(h);
        Ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += world->getSceneColor(r, &local_rand_state);
    }

    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(Object** aux, int numobjects, SceneGPU* d_world, Camera** d_camera, int nx, int ny, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_world->setSkyColor(Vec3(0.5f, 0.7f, 1.0f));
        d_world->setInfColor(Vec3(1.0f, 1.0f, 1.0f));
        d_world->setList(aux, numobjects);

        d_world->add(new Object(
            new Sphere(Vec3(0.0f, -1000.0f, 0.0f), 1000.0f),
            new Diffuse(Vec3(0.5f, 0.5f, 0.5f))
        ));
        d_world->add(new Object(
            new Sphere(Vec3(4.0f, 1.0f, 0.0f), 1.0f),
            new Metallic(Vec3(0.7f, 0.6f, 0.5f), 0.0f)
        ));
        d_world->add(new Object(
            new Sphere(Vec3(-4.0f, 1.0f, 0.0f), 1.0f),
            new Diffuse(Vec3(0.4f, 0.2f, 0.1f))
        ));
        d_world->add(new Object(
            new Sphere(Vec3(0.0f, 1.0f, 0.0f), 1.0f),
            new Crystalline(1.5f)
        ));
        /*
        Object Sphere ( (0.0, -1000.0, 0.0), 1000.0 ) Diffuse ( (0.5, 0.5, 0.5) )
        Object Sphere ( (4.0, 1.0, 0.0), 1.0 ) Metallic ( (0.7, 0.6, 0.5), 0.0 )
        Object Sphere ( (-4.0, 1.0, 0.0), 1.0 ) Diffuse ( (0.4, 0.2, 0.1) )
        Object Sphere ( (0.0, 1.0, 0.0), 1.0 ) Crystalline ( 1.5 )
        
        
        d_world->add(new Object(
            new Sphere(Vec3(0.0f, -1000.0f, 0.0f), 1000.0f),
            new Diffuse(Vec3(0.5f, 0.5f, 0.5f))
        ));
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                Vec3 center(a + RND, 0.2f, b + RND);
                if (choose_mat < 0.8f) {
                    d_world->add(new Object(
                        new Sphere(center, 0.2f),
                        new Diffuse(Vec3(RND * RND,
                            RND * RND,
                            RND * RND))
                    ));
                } else if (choose_mat < 0.95f) {
                    d_world->add(new Object(
                        new Sphere(center, 0.2f),
                        new Metallic(Vec3(0.5f * (1.0f + RND),
                            0.5f * (1.0f + RND),
                            0.5f * (1.0f + RND)),
                            0.5f * RND)
                    ));
                } else {
                    d_world->add(new Object(
                        new Sphere(center, 0.2f),
                        new Crystalline(1.5f)
                    ));
                }
            }
        }
        d_world->add(new Object(
            new Sphere(Vec3(0.0f, 1.0f, 0.0f), 1.0f),
            new Crystalline(1.5f)
        ));
        d_world->add(new Object(
            new Sphere(Vec3(-4.0f, 1.0f, 0.0f), 1.0f),
            new Diffuse(Vec3(0.4f, 0.2f, 0.1f))
        ));
        d_world->add(new Object(
            new Sphere(Vec3(4.0f, 1.0f, 0.0f), 1.0f),
            new Metallic(Vec3(0.7f, 0.6f, 0.5f), 0.0f)
        ));
        */
        *rand_state = local_rand_state;

        Vec3 lookfrom(13.0f, 2.0f, 3.0f);
        Vec3 lookat(0.0f, 0.0f, 0.0f);
        float dist_to_focus = 10.0f; //(lookfrom - lookat).length();
        float aperture = 0.1f;
        *d_camera = new Camera(lookfrom,
            lookat,
            Vec3(0.0f, 1.0f, 0.0f),
            20.0,
            float(nx) / float(ny),
            aperture,
            dist_to_focus);
    }
}

void rayTracingGPU(Vec3* img, int w, int h, int ns = 1, int tx = 8, int ty = 8) {
    //int tx = 8;
    //int ty = 8;
    clock_t start, stop;
    double timer_seconds;

    //std::cerr << "Rendering a " << w << "x" << h << " image with " << ns << " samples per pixel ";
    //std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = w * h;
    size_t fb_size = num_pixels * sizeof(Vec3);

    start = clock();
    // allocate FB
    Vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init << <1, 1 >> > (d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    Object** aux;
    int numobjects = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void**)&aux, numobjects * sizeof(Object*)));
    SceneGPU* d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Scene)));
    Camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(Camera*)));
    create_world << <1, 1 >> > (aux, numobjects, d_world, d_camera, w, h, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    //std::cerr << "Loading took " << timer_seconds << " seconds.\n";

    start = clock();
    // Render our buffer
    dim3 blocks(w / tx + 1, h / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (w, h, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render << <blocks, threads >> > (fb, w, h, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    //std::cerr << "Rendering took " << timer_seconds << " seconds.\n";


    // Output FB as Image
    start = clock();
    for (int i = h - 1; i >= 0; i--) {
        for (int j = 0; j < w; j++) {
            size_t pixel_index = i * w + j;
            img[pixel_index] = fb[pixel_index];
        }
    }
    stop = clock();
    timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    //std::cerr << "Copy took " << timer_seconds << " seconds.\n";

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(aux));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
}
